#include "hip/hip_runtime.h"
#include <fstream>
#include <getopt.h>
#include <inttypes.h>
#include <iostream>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include "crispacuda.h"

char* bits_to_string(uint64_t text, uint64_t length) {
    char *s = (char*)malloc(length + 1);
    memset(s, 0, length + 1);
    uint64_t shift = 2 * ( length - 1 ); //there are twice as many bits as there are characters

    //fill with N if its an error string (all bits set to 1)
    if ( text == ERROR_STR ) {
        memset(s, 'N', length);
    }

    //extract each character from the text
    for ( int i = 0; i < length; i++, shift -= 2 ) {
        //put the character we're interested in at the very end
        //of the integer, and switch all remaining bits to 0 with & 0x3
        uint8_t character = (text >> shift) & 0x3;
        switch ( character ) {
            case 0: s[i] = 'A'; break;
            case 1: s[i] = 'C'; break;
            case 2: s[i] = 'G'; break;
            case 3: s[i] = 'T'; break;
            default: break;
        }
    }

    return s;
}

void print_seq(uint64_t text, uint64_t length) {
    char *seq = bits_to_string(text, length);
    printf("%s\n", seq);
    free(seq);
}

uint64_t revcom(uint64_t text, int length) {
    unsigned int num_bits = sizeof(text) * CHAR_BIT;

    uint64_t mask = 0xFFFFFFFFFFFFFFFFull >> ( (num_bits - (length * 2)) - 1 );
    text = ~text & mask;
    uint64_t reversed = text >> (length * 2);
    int shift = 0;

    for ( int i = 0; i < length; i++, shift += 2 ) {
        reversed <<= 2;
        reversed |= ( text >> shift ) & 0x3;
    }
    return reversed;
}

__device__ targets_t targets;

__device__ void push_back(uint64_t id, int mm) {
    int insert_pt = atomicAdd(&targets.offc, 1);
    if ( insert_pt < max_off_list ) {
        targets.off[insert_pt] = id;
    }
    if ( mm == 0 ) {
        insert_pt = atomicAdd(&targets.onc, 1);
        if ( insert_pt < max_on_list ) {
            targets.on[insert_pt] = id;
        }
    }
}

__global__
void find_off_targets(uint64_t *crisprs, crispr_t query, int *summary, metadata_t metadata) {
    const uint64_t pam_on = 0x1ull << metadata.seq_length *2;
    const uint64_t pam_off = ~pam_on;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    if(blockIdx.x == 0 && threadIdx.x == 0) {
        targets.offc = 0;
        targets.onc  = 0;
    }
    __syncthreads();
    for ( uint64_t j = index; j < metadata.num_seqs; j+= stride ) {
        uint64_t test_crispr = crisprs[j];
        if ( test_crispr == ERROR_STR ) continue;

        uint64_t match = query.seq ^ test_crispr;
        if ( match & pam_on ) {
            match = query.rev_seq ^ test_crispr;
        }

        match = match & pam_off;
        match = (match | (match >> 1)) & (0x5555555555555555ull);
        int mm = __popcll(match);

        if ( mm <= max_mismatches ) {
            atomicAdd(&summary[mm], 1);
            push_back(j + 1 + metadata.offset, mm);
        }
    }
}

void write_output(FILE *fp, crispr_t query, targets_t targets, int *summary,
        int species_id, bool store_offs) {
    int onc = std::min(targets.onc, max_on_list);
    int offc = std::min(targets.offc, max_off_list);
    if ( fp != NULL ) {
        fwrite(&query.id, sizeof(uint64_t), 1, fp); 
        fwrite(summary, sizeof(int), max_mismatches+1, fp);
        fwrite(&onc, sizeof(int), 1, fp);
        fwrite(&offc, sizeof(int), 1, fp);
        fwrite(targets.on, sizeof(uint64_t), onc, fp);
        fwrite(targets.off, sizeof(uint64_t), offc, fp);
    } else {
        thrust::sort(thrust::host, targets.off, targets.off + offc, thrust::less<uint64_t>());
        std::cout << query.id << "\t" << int(species_id);
        const char *sep = seps[0];
        if(!store_offs || targets.offc > max_off_list) {
            std::cout << "\t\\N\t{";
        } else {
            std::cout << "\t{";
            for( int j = 0; j < offc; j++ ) {
                std::cout << sep << targets.off[j];
                sep = seps[1];
            }
            std::cout << "}\t{";
        }
        sep = seps[0];
        for( int j = 0; j <= max_mismatches; j++ ) {
            std::cout << sep << j << ": " << summary[j];
            sep = seps[2];
        }
        std::cout << "}" << std::endl;
    }
}

void calc_off_targets(FILE *fp, uint64_t *crisprs, crispr_t query, metadata_t metadata,
        options_t options) {
    int summary_size = (max_mismatches+1)*sizeof(int);
    int *summary;
    hipMalloc((void**)&summary, summary_size);
    hipMemset(summary, 0, summary_size);

    const int blockSize = 128;
    const int numBlocks = (metadata.num_seqs + blockSize - 1) / blockSize;
    find_off_targets<<<numBlocks, blockSize>>>(crisprs, query, summary, metadata); 
    hipDeviceSynchronize();

    targets_t h_targets;
    int *h_summary = (int*)malloc(summary_size);
    hipMemcpyFromSymbol(&h_targets, HIP_SYMBOL(targets), sizeof(targets_t));
    hipMemcpy(h_summary, summary, summary_size, hipMemcpyDeviceToHost);
    
    write_output(fp, query, h_targets, h_summary, metadata.species_id, options.store_offs);

    hipFree(summary);
    free(h_summary);
}

metadata_t load_metadata(FILE *fp) {
    uint8_t endian_test;
    fread(&endian_test, sizeof(uint8_t), 1, fp);
    if ( endian_test != 1 ) {
        fprintf(stderr, "Endianess of the file does not match your hardware\n");
        exit(1);
    }

    uint32_t file_version;
    fread(&file_version, sizeof(uint32_t), 1, fp);
    if ( file_version != VERSION ) {
        fprintf(stderr, "Index file is the wrong version! Please regenerate!\n");
        exit(1);
    }
    fprintf(stderr, "Version is %d\n", file_version);

    metadata_t metadata;
    fread(&metadata, sizeof(metadata_t), 1, fp);
    fprintf(stderr, "Assembly is %s (%s)\n", metadata.assembly, metadata.species);
    fprintf(stderr, "File has %" PRIu64 " sequences\n", metadata.num_seqs);
    fprintf(stderr, "Sequence length is %" PRIu64 "\n", metadata.seq_length);
    fprintf(stderr, "Offset is %" PRIu64 "\n", metadata.offset);
    fprintf(stderr, "Species id is %d\n", metadata.species_id);
    return metadata;
}

uint64_t read_options(int argc, char *argv[], search_t *search, options_t *options) {
    int c = -1;
    uint64_t start = 0, num = 0;
    while ( ( c = getopt(argc, argv, "s:n:i:o:qm:") ) != -1 )  {
        switch(c) {
            case 's': start = atol(optarg); break;
            case 'n': num   = atol(optarg); break;
            case 'i': (*search).index_file = optarg; break;
            case 'o': (*search).output_file = optarg; break;
            case 'q': (*options).store_offs = false; break;
            case 'm': (*options).max_mismatches = atoi(optarg); break;

        }
    }
    if ( (*search).index_file == NULL ) {
        fprintf(stderr, "An index file must be specified with the -i option\n");
        return 0;
    }
    if ( start != 0 && num == 0 ) {
        fprintf(stderr, "If -s is specified, -n must be also");
        return 0;
    }
    if ( start == 0 && num != 0 ) {
        fprintf(stderr, "If -n is specified, -s must be also");
        return 0;
    }
    uint64_t num_queries = argc - optind + num;
    (*search).queries = (crispr_t*)malloc(num_queries * sizeof(crispr_t));
    for ( int i = 0; i < num; i++ ) {
        (*search).queries[i].id = start + i;
    }
    for ( int i = optind; i < argc; i++ ) {
        uint64_t id = atol(argv[i]);
        if ( id == 0 ) {
            fprintf(stderr, "Could not parse '%s' an ID\n", argv[i]);
            return false;
        }
        (*search).queries[i - optind + num].id = id;
    }
    return num_queries;
}

int main(int argc, char *argv[]) {
    search_t search = default_search;
    options_t options = default_options;
    uint64_t num_queries = read_options(argc, argv, &search, &options);
    if ( num_queries == 0 ) {
        return 1;
    }

    FILE *fp = fopen(search.index_file, "r");
    if ( fp == NULL ) {
        fprintf(stderr, "Could not open index\n");
        exit(1);
    }
    metadata_t metadata = load_metadata(fp);
    clock_t t = clock();
    const uint64_t data_size = metadata.num_seqs * sizeof(uint64_t);
    uint64_t *h_crisprs = (uint64_t*)malloc(data_size);
    fread(h_crisprs, sizeof(uint64_t), metadata.num_seqs, fp);
    t = clock() - t;
    fclose(fp);
    fprintf(stderr, "Loading took %f seconds\n", ((float)t)/CLOCKS_PER_SEC);

    for ( int i = 0; i < num_queries; i++ ) {
        if ( search.queries[i].id < metadata.offset + 1
                || search.queries[i].id > metadata.offset + metadata.num_seqs ) {
            fprintf(stderr, "%" PRIu64 " is not a valid ID in this index\n",
                    search.queries[i].id);
            return 2;
        }
        search.queries[i].seq = h_crisprs[search.queries[i].id - metadata.offset - 1];
        search.queries[i].rev_seq = revcom(search.queries[i].seq, metadata.seq_length);
    }

    uint64_t *crisprs;
    hipMalloc((void**)&crisprs, data_size);
    hipMemcpy(crisprs, h_crisprs, data_size, hipMemcpyHostToDevice);
    free(h_crisprs);

    FILE *fw = NULL;
    if ( search.output_file != NULL ) {
        fw = fopen(search.output_file, "w");
        if ( fw == NULL ) {
            fprintf(stderr, "Could not open output file\n");
            exit(1);
        }
    }
    for ( int i = 0; i < num_queries; i++ ) {
        calc_off_targets(fw, crisprs, search.queries[i], metadata, options);
    }
    if ( fw != NULL ) {
        fclose(fw);
    }

    hipFree(crisprs);
    free(search.queries);
}
