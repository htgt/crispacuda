#include "hip/hip_runtime.h"
#include <fstream>
#include <getopt.h>
#include <inttypes.h>
#include <iostream>
#include <ostream>
#include <sstream>
#include <iomanip>
#include <string.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <vector>
#include "crispacuda.h"
#include "mongoose.h"
#include "options.h"
#include "seq.h"

uint64_t *crisprs;
uint64_t *h_crisprs;
metadata_t metadata;
options_t options;
__device__ targets_t targets;
__constant__ __device__ uint64_t pam_on;
__constant__ __device__ uint64_t pam_off;
__constant__ __device__ metadata_t d_metadata;
__constant__ __device__ options_t d_options;

__device__ void push_back(uint64_t id, int mm) {
    int insert_pt = atomicAdd(&targets.offc, 1);
    if ( insert_pt < max_off_list ) {
        targets.off[insert_pt] = id;
    }
    if ( mm == 0 ) {
        insert_pt = atomicAdd(&targets.onc, 1);
        if ( insert_pt < max_on_list ) {
            targets.on[insert_pt] = id;
        }
    }
}

__global__
void find_off_targets(uint64_t *crisprs, crispr_t query, int *summary) {
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    int stride = blockDim.x * gridDim.x;
    if(blockIdx.x == 0 && tid <= d_options.max_mismatches) {
        targets.offc = 0;
        targets.onc  = 0;
        summary[tid] = 0;
    }
    __syncthreads();
    for ( uint64_t j = index; j < d_metadata.num_seqs; j+= stride ) {
        uint64_t test_crispr = crisprs[j];
        if ( test_crispr == ERROR_STR ) continue;

        uint64_t match = query.seq ^ test_crispr;
        if ( match & pam_on ) {
            match = query.rev_seq ^ test_crispr;
        }

        match = match & pam_off;
        match = (match | (match >> 1)) & (0x5555555555555555ull);
        int mm = __popcll(match);

        if ( mm <= d_options.max_mismatches ) {
            atomicAdd(&summary[mm], 1);
            push_back(j + 1 + d_metadata.offset, mm);
        }
    }
}

void write_output(std::ostream &stream, crispr_t query, int *summary, targets_t targets) {
    int onc = std::min(targets.onc, max_on_list);
    int offc = std::min(targets.offc, max_off_list);
    thrust::sort(thrust::host, targets.off, targets.off + offc, thrust::less<uint64_t>());
    stream << query.id << "\t" << int(metadata.species_id);
    const char *sep = seps[0];
    if(!options.store_offs || targets.offc > max_off_list) {
        stream << "\t\\N\t{";
    } else {
        stream << "\t{";
        for( int j = 0; j < offc; j++ ) {
            stream << sep << targets.off[j];
            sep = seps[1];
        }
        stream << "}\t{";
    }
    sep = seps[0];
    for( int j = 0; j <= options.max_mismatches; j++ ) {
        stream << sep << j << ": " << summary[j];
        sep = seps[2];
    }
    stream << "}" << std::endl;
}

void calc_off_targets(std::ostream &stream, crispr_t query) {
    int summary_size = (max_mismatches + 1) * sizeof(int);
    int *summary;
    hipMalloc((void**)&summary, summary_size);
    const int blockSize = 128;
    const int numBlocks = (metadata.num_seqs + blockSize - 1) / blockSize;
    find_off_targets<<<numBlocks, blockSize>>>(crisprs, query, summary); 
    hipDeviceSynchronize();

    targets_t h_targets;
    int *h_summary = (int*)malloc(summary_size);
    hipMemcpyFromSymbol(&h_targets, HIP_SYMBOL(targets), sizeof(targets_t), 0, hipMemcpyDeviceToHost);
    hipMemcpy(h_summary, summary, summary_size, hipMemcpyDeviceToHost);
    
    write_output(stream, query, h_summary, h_targets);

    free(h_summary);
}

metadata_t load_metadata(FILE *fp) {
    uint8_t endian_test;
    CHECK_FREAD(&endian_test, sizeof(uint8_t), 1, fp);
    if ( endian_test != 1 ) {
        fprintf(stderr, "Endianess of the file does not match your hardware\n");
        exit(1);
    }

    uint32_t file_version;
    CHECK_FREAD(&file_version, sizeof(uint32_t), 1, fp);
    if ( file_version != VERSION ) {
        fprintf(stderr, "Index file is the wrong version! Please regenerate!\n");
        exit(1);
    }
    fprintf(stderr, "Version is %d\n", file_version);

    metadata_t metadata;
    CHECK_FREAD(&metadata, sizeof(metadata_t), 1, fp);
    fprintf(stderr, "Assembly is %s (%s)\n", metadata.assembly, metadata.species);
    fprintf(stderr, "File has %" PRIu64 " sequences\n", metadata.num_seqs);
    fprintf(stderr, "Sequence length is %" PRIu64 "\n", metadata.seq_length);
    fprintf(stderr, "Offset is %" PRIu64 "\n", metadata.offset);
    fprintf(stderr, "Species id is %d\n", metadata.species_id);
    return metadata;
}

void parse_request(struct mg_str req, std::vector<uint64_t> &ids) {
    char *query, *id;
    query = (char*)malloc(req.len + 1);
    memset(query, 0, req.len + 1);
    strncpy(query, req.p, req.len);
    while ( (id = strsep(&query, "\n")) != NULL ) {
        ids.push_back(atol(id));
    }
}

static void handle_request(struct mg_connection *c, int ev, void *p) {
    if ( ev == MG_EV_HTTP_REQUEST ) {
        struct http_message *hm = (struct http_message *)p;
        
        if( mg_vcmp(&hm->uri, "/search") == 0 ) {
            std::vector<uint64_t> ids;
            parse_request(hm->body, ids);
            std::stringstream results;
            for(uint64_t id : ids) {
                crispr_t crispr;
                if ( id <= metadata.offset || id > metadata.offset + metadata.num_seqs ) {
                    continue;
                }
                crispr.id = id;
                crispr.seq = h_crisprs[id - metadata.offset - 1];
                crispr.rev_seq = revcom(crispr.seq, metadata.seq_length);
                calc_off_targets(results, crispr);
            }
            const std::string tmp = results.str();
            struct mg_str response = mg_mk_str(tmp.c_str());
            mg_send_head(c, 200, response.len, "Content-Type: text/plain");
            mg_printf(c, "%.*s", (int)response.len, response.p);
        } else if( mg_vcmp(&hm->uri, "/favicon.ico") == 0 ) {
            mg_http_serve_file(c, hm, "favicon.ico", mg_mk_str("image/ico"), mg_mk_str(""));
        } else {
            mg_http_serve_file(c, hm, "index.htm", mg_mk_str("text/html"), mg_mk_str(""));
        }
    }
}

void run_server(char *port) {
    struct mg_mgr mgr;
    struct mg_connection *c;
    mg_mgr_init(&mgr, NULL);
    c = mg_bind(&mgr, port, handle_request);
    mg_set_protocol_http_websocket(c);
    for(;;) {
        mg_mgr_poll(&mgr, 1000000);
    }
    mg_mgr_free(&mgr);
}

int main(int argc, char *argv[]) {
    populate_cmap();
    search_t search = default_search;
    options = default_options;
    int64_t num_queries = read_options(argc, argv, &search, &options);
    if ( num_queries <= 0  && search.port == NULL) {
        return num_queries;
    }

    FILE *fp = fopen(search.index_file, "r");
    if ( fp == NULL ) {
        fprintf(stderr, "Could not open index\n");
        exit(1);
    }
    metadata = load_metadata(fp);
    clock_t t = clock();
    const uint64_t data_size = metadata.num_seqs * sizeof(uint64_t);
    h_crisprs = (uint64_t*)malloc(data_size);
    CHECK_FREAD(h_crisprs, sizeof(uint64_t), metadata.num_seqs, fp);
    t = clock() - t;
    fclose(fp);
    fprintf(stderr, "Loading took %f seconds\n", ((float)t)/CLOCKS_PER_SEC);
    
    uint64_t h_pam_on = 0x1ull << metadata.seq_length *2;
    uint64_t h_pam_off = ~h_pam_on;
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(pam_on), &h_pam_on, sizeof(uint64_t)));
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(pam_off), &h_pam_off, sizeof(uint64_t)));
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_metadata), &metadata, sizeof(metadata_t)));
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_options), &options, sizeof(options_t)));

    for ( int i = 0; i < num_queries; i++ ) {
        if ( search.search_by_seq ) {
            search.queries[i].seq = string_to_bits(argv[search.queries[i].id], metadata.seq_length, 1);
            if ( search.queries[i].seq == ERROR_STR ) {
                fprintf(stderr, "%s is not a valid sequence\n", argv[search.queries[i].id]);
                return 2;
            }
            search.queries[i].id = 0;
        } else {
            if ( search.queries[i].id < metadata.offset + 1
                    || search.queries[i].id > metadata.offset + metadata.num_seqs ) {
                fprintf(stderr, "%" PRIu64 " is not a valid ID in this index\n", search.queries[i].id);
                return 2;
            }
            search.queries[i].seq = h_crisprs[search.queries[i].id - metadata.offset - 1];
        }
        search.queries[i].rev_seq = revcom(search.queries[i].seq, metadata.seq_length);
    }

    size_t free_memory, total_memory;
    hipMemGetInfo(&free_memory, &total_memory);
    fprintf(stderr, "Requires %" PRIu64 "mb of GPU memory, %" PRIu64 "mb is available\n",
            data_size >> 20, free_memory >> 20);
    if ( data_size > free_memory ) {
        fprintf(stderr, "Insufficient GPU memory, exiting.\n");
        return 3;
    }
    CHECK_CUDA(hipMalloc((void**)&crisprs, data_size));
    CHECK_CUDA(hipMemcpy(crisprs, h_crisprs, data_size, hipMemcpyHostToDevice));

    for ( int i = 0; i < num_queries; i++ ) {
        calc_off_targets(std::cout, search.queries[i]);
    }
    if ( search.port != NULL ) {
        printf("Starting server on port %s...\n", search.port);
        run_server(search.port);
    }

    free(h_crisprs);
    hipFree(crisprs);
    free(search.queries);
    return 0;
}
