#include "hip/hip_runtime.h"
#include <fstream>
#include <getopt.h>
#include <inttypes.h>
#include <iostream>
#include <ostream>
#include <sstream>
#include <iomanip>
#include <string.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <vector>
#include "crispacuda.h"
#include "devices.h"
#include "mongoose.h"
#include "seq.h"

uint64_t revcom(uint64_t text, int length) {
    unsigned int num_bits = sizeof(text) * CHAR_BIT;

    uint64_t mask = 0xFFFFFFFFFFFFFFFFull >> ( (num_bits - (length * 2)) - 1 );
    text = ~text & mask;
    uint64_t reversed = text >> (length * 2);
    int shift = 0;

    for ( int i = 0; i < length; i++, shift += 2 ) {
        reversed <<= 2;
        reversed |= ( text >> shift ) & 0x3;
    }
    return reversed;
}

uint64_t *crisprs;
uint64_t *h_crisprs;
metadata_t metadata;
options_t options;
__device__ targets_t targets;
__constant__ __device__ uint64_t pam_on;
__constant__ __device__ uint64_t pam_off;
__constant__ __device__ metadata_t d_metadata;
__constant__ __device__ options_t d_options;

__device__ void push_back(uint64_t id, int mm) {
    int insert_pt = atomicAdd(&targets.offc, 1);
    if ( insert_pt < max_off_list ) {
        targets.off[insert_pt] = id;
    }
    if ( mm == 0 ) {
        insert_pt = atomicAdd(&targets.onc, 1);
        if ( insert_pt < max_on_list ) {
            targets.on[insert_pt] = id;
        }
    }
}

__global__
void find_off_targets(uint64_t *crisprs, crispr_t query, int *summary) {
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    int stride = blockDim.x * gridDim.x;
    if(blockIdx.x == 0 && tid == 0 ) {
        targets.offc = 0;
        targets.onc  = 0;
    }
    __syncthreads();
    for ( uint64_t j = index; j < d_metadata.num_seqs; j+= stride ) {
        uint64_t test_crispr = crisprs[j];
        if ( test_crispr == ERROR_STR ) continue;

        uint64_t match = query.seq ^ test_crispr;
        if ( match & pam_on ) {
            match = query.rev_seq ^ test_crispr;
        }

        match = match & pam_off;
        match = (match | (match >> 1)) & (0x5555555555555555ull);
        int mm = __popcll(match);

        if ( mm <= d_options.max_mismatches ) {
            atomicAdd(&summary[mm], 1);
            push_back(j + 1 + d_metadata.offset, mm);
        }
    }
}

void write_output(std::ostream &stream, crispr_t query, int *summary, targets_t targets) {
    int onc = std::min(targets.onc, max_on_list);
    int offc = std::min(targets.offc, max_off_list);
    thrust::sort(thrust::host, targets.off, targets.off + offc, thrust::less<uint64_t>());
    stream << query.id << "\t" << int(metadata.species_id);
    const char *sep = seps[0];
    if(!options.store_offs || targets.offc > max_off_list) {
        stream << "\t\\N\t{";
    } else {
        stream << "\t{";
        for( int j = 0; j < offc; j++ ) {
            stream << sep << targets.off[j];
            sep = seps[1];
        }
        stream << "}\t{";
    }
    sep = seps[0];
    for( int j = 0; j <= options.max_mismatches; j++ ) {
        stream << sep << j << ": " << summary[j];
        sep = seps[2];
    }
    stream << "}" << std::endl;
}

void calc_off_targets(std::ostream &stream, crispr_t query) {
    int summary_size = (options.max_mismatches + 1) * sizeof(int);
    int *summary;
    hipMalloc((void**)&summary, summary_size);
    hipMemset(summary, 0, options.max_mismatches);
    const int blockSize = 128;
    const int numBlocks = (metadata.num_seqs + blockSize - 1) / blockSize;
    find_off_targets<<<numBlocks, blockSize>>>(crisprs, query, summary); 
    hipDeviceSynchronize();

    targets_t h_targets;
    int *h_summary = (int*)malloc(summary_size);
    hipMemcpyFromSymbol(&h_targets, HIP_SYMBOL(targets), sizeof(targets_t), 0, hipMemcpyDeviceToHost);
    hipMemcpy(h_summary, summary, summary_size, hipMemcpyDeviceToHost);
    
    write_output(stream, query, h_summary, h_targets);

    free(h_summary);
}

metadata_t load_metadata(FILE *fp) {
    uint8_t endian_test;
    CHECK_FREAD(&endian_test, sizeof(uint8_t), 1, fp);
    if ( endian_test != 1 ) {
        fprintf(stderr, "Endianess of the file does not match your hardware\n");
        exit(1);
    }

    uint32_t file_version;
    CHECK_FREAD(&file_version, sizeof(uint32_t), 1, fp);
    if ( file_version != VERSION ) {
        fprintf(stderr, "Index file is the wrong version! Please regenerate!\n");
        exit(1);
    }
    fprintf(stderr, "Version is %d\n", file_version);

    metadata_t metadata;
    CHECK_FREAD(&metadata, sizeof(metadata_t), 1, fp);
    fprintf(stderr, "Assembly is %s (%s)\n", metadata.assembly, metadata.species);
    fprintf(stderr, "File has %" PRIu64 " sequences\n", metadata.num_seqs);
    fprintf(stderr, "Sequence length is %" PRIu64 "\n", metadata.seq_length);
    fprintf(stderr, "Offset is %" PRIu64 "\n", metadata.offset);
    fprintf(stderr, "Species id is %d\n", metadata.species_id);
    return metadata;
}

int64_t read_options(int argc, char *argv[], search_t *search, options_t *options) {
    int c = -1, device = 0;
    uint64_t start = 0, num = 0;
    bool show_help = false;
    while ( ( c = getopt(argc, argv, "s:n:i:m:d:p:hqz") ) != -1 )  {
        switch(c) {
            case 's': start = atol(optarg); break;
            case 'n': num   = atol(optarg); break;
            case 'i': (*search).index_file = optarg; break;
            case 'q': (*options).store_offs = false; break;
            case 'm': (*options).max_mismatches = atoi(optarg); break;
            case 'h': show_help = true; break;
            case 'd': device = atoi(optarg); break;
            case 'z': (*search).search_by_seq = true; break;
            case 'p': (*search).port = optarg; break;
        }
    }
    if ( show_help ) {
        printf("CRISPACUDA\n");
        printf("Searches for CRISPR off-targets on a GPU\n");
        printf("Usage: crispacuda [options] <ids...>\n");
        printf("Contact: Joel Rein joel.rein@sanger.ac.uk\n\n");
        printf("OPTIONS:\n");
        printf("-i <FILE>\n");
        printf("\tSpecify the index file to search. REQUIRED.\n");
        printf("-s <INT>\n");
        printf("\tThe index to start searches from\n");
        printf("-n <INT>\n");
        printf("\tHow many off-targets to calculate from start.\n");
        printf("\tRequired if, and only if, -s is specified.\n");
        printf("-m <INT>\n");
        printf("\tThe maximum number of mismatches to record results for. Defaults to 4.\n");
        printf("-q\n\tDo not report a list of off-targets.\n");
        printf("\t Has no effect if outputting to a binary file.\n");
        printf("-h\n\tPrint this help message and GPU information\n");
        printf("-d <INT>\n\tThe GPU device to use.\n");
        printf("-z\n\tSpecify CRISPRs as strings rather than IDs\n");
        printf("-p\n\tStart a webserver on port <number>\n");
        printf("Following these arguments you may specify individual CRISPRs to search.\n\n");
        printf("EXIT CODES\n");
        printf("\tNegative exit codes indicate errors in command line options.\n");
        printf("\tPositive exit codes indicate errors running the search.\n");
        printf("\tReturns 0 on success.\n\n");
        show_devices();
        return 0;
    }
    if ( device >= 1 && hipSetDevice(device) != hipSuccess ) {
        fprintf(stderr, "Could not use device %d\n", device);
        return -1;
    }
    if ( (*search).index_file == NULL ) {
        fprintf(stderr, "An index file must be specified with the -i option\n");
        return -2;
    }
    if ( start != 0 && num == 0 ) {
        fprintf(stderr, "If -s is specified, -n must be also");
        return -3;
    }
    if ( start == 0 && num != 0 ) {
        fprintf(stderr, "If -n is specified, -s must be also");
        return -4;
    }
    if ( start != 0 && (*search).search_by_seq ) {
        fprintf(stderr, "-s and -n are not compatible with search by sequence\n");
        return -5;
    }
    int64_t num_queries = argc - optind + num;
    (*search).queries = (crispr_t*)malloc(num_queries * sizeof(crispr_t));
    memset((*search).queries, 0, num_queries * sizeof(crispr_t));
    for ( int i = 0; i < num; i++ ) {
        (*search).queries[i].id = start + i;
    }
    for ( int i = optind; i < argc; i++ ) {
        uint64_t id = 0;
        if ((*search).search_by_seq) {
            id = i;
        } else {
            id = atol(argv[i]);
            if ( id == 0 ) {
                fprintf(stderr, "Could not parse '%s' an ID\n", argv[i]);
                return -6;
            }
        }
        (*search).queries[i - optind + num].id = id;
    }
    return num_queries;
}

void parse_request(struct mg_str req, std::vector<uint64_t> &ids) {
    char *query, *id;
    query = (char*)malloc(req.len + 1);
    memset(query, 0, req.len + 1);
    strncpy(query, req.p, req.len);
    while ( (id = strsep(&query, "\n")) != NULL ) {
        ids.push_back(atol(id));
    }
}

static void handle_request(struct mg_connection *c, int ev, void *p) {
    if ( ev == MG_EV_HTTP_REQUEST ) {
        struct http_message *hm = (struct http_message *)p;
        
        if( mg_vcmp(&hm->uri, "/search") == 0 ) {
            std::vector<uint64_t> ids;
            parse_request(hm->body, ids);
            std::stringstream results;
            for(uint64_t id : ids) {
                crispr_t crispr;
                if ( id <= metadata.offset || id > metadata.offset + metadata.num_seqs ) {
                    continue;
                }
                crispr.id = id;
                crispr.seq = h_crisprs[id - metadata.offset - 1];
                crispr.rev_seq = revcom(crispr.seq, metadata.seq_length);
                calc_off_targets(results, crispr);
            }
            const std::string tmp = results.str();
            struct mg_str response = mg_mk_str(tmp.c_str());
            mg_send_head(c, 200, response.len, "Content-Type: text/plain");
            mg_printf(c, "%.*s", (int)response.len, response.p);
        } else if( mg_vcmp(&hm->uri, "/favicon.ico") == 0 ) {
            mg_http_serve_file(c, hm, "favicon.ico", mg_mk_str("image/ico"), mg_mk_str(""));
        } else {
            mg_http_serve_file(c, hm, "index.htm", mg_mk_str("text/html"), mg_mk_str(""));
        }
    }
}

void run_server(char *port) {
    struct mg_mgr mgr;
    struct mg_connection *c;
    mg_mgr_init(&mgr, NULL);
    c = mg_bind(&mgr, port, handle_request);
    mg_set_protocol_http_websocket(c);
    for(;;) {
        mg_mgr_poll(&mgr, 1000000);
    }
    mg_mgr_free(&mgr);
}

int main(int argc, char *argv[]) {
    populate_cmap();
    search_t search = default_search;
    options = default_options;
    int64_t num_queries = read_options(argc, argv, &search, &options);
    if ( num_queries <= 0  && search.port == NULL) {
        return num_queries;
    }

    FILE *fp = fopen(search.index_file, "r");
    if ( fp == NULL ) {
        fprintf(stderr, "Could not open index\n");
        exit(1);
    }
    metadata = load_metadata(fp);
    clock_t t = clock();
    const uint64_t data_size = metadata.num_seqs * sizeof(uint64_t);
    h_crisprs = (uint64_t*)malloc(data_size);
    CHECK_FREAD(h_crisprs, sizeof(uint64_t), metadata.num_seqs, fp);
    t = clock() - t;
    fclose(fp);
    fprintf(stderr, "Loading took %f seconds\n", ((float)t)/CLOCKS_PER_SEC);
    
    uint64_t h_pam_on = 0x1ull << metadata.seq_length *2;
    uint64_t h_pam_off = ~h_pam_on;
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(pam_on), &h_pam_on, sizeof(uint64_t)));
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(pam_off), &h_pam_off, sizeof(uint64_t)));
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_metadata), &metadata, sizeof(metadata_t)));
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_options), &options, sizeof(options_t)));

    for ( int i = 0; i < num_queries; i++ ) {
        if ( search.search_by_seq ) {
            search.queries[i].seq = string_to_bits(argv[search.queries[i].id], metadata.seq_length, 1);
            if ( search.queries[i].seq == ERROR_STR ) {
                fprintf(stderr, "%s is not a valid sequence\n", argv[search.queries[i].id]);
                return 2;
            }
            search.queries[i].id = 0;
        } else {
            if ( search.queries[i].id < metadata.offset + 1
                    || search.queries[i].id > metadata.offset + metadata.num_seqs ) {
                fprintf(stderr, "%" PRIu64 " is not a valid ID in this index\n", search.queries[i].id);
                return 2;
            }
            search.queries[i].seq = h_crisprs[search.queries[i].id - metadata.offset - 1];
        }
        search.queries[i].rev_seq = revcom(search.queries[i].seq, metadata.seq_length);
    }

    size_t free_memory, total_memory;
    hipMemGetInfo(&free_memory, &total_memory);
    fprintf(stderr, "Requires %" PRIu64 "mb of GPU memory, %" PRIu64 "mb is available\n",
            data_size >> 20, free_memory >> 20);
    if ( data_size > free_memory ) {
        fprintf(stderr, "Insufficient GPU memory, exiting.\n");
        return 3;
    }
    CHECK_CUDA(hipMalloc((void**)&crisprs, data_size));
    CHECK_CUDA(hipMemcpy(crisprs, h_crisprs, data_size, hipMemcpyHostToDevice));

    for ( int i = 0; i < num_queries; i++ ) {
        calc_off_targets(std::cout, search.queries[i]);
    }
    if ( search.port != NULL ) {
        printf("Starting server on port %s...\n", search.port);
        run_server(search.port);
    }

    free(h_crisprs);
    hipFree(crisprs);
    free(search.queries);
    return 0;
}
